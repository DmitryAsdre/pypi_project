#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <iostream>
#include "_test_reduction.hh"

__global__ void calculate_sum_cu_device(double * data, size_t size, volatile double * res){
    int gid = threadIdx.x + blockDim.x*blockIdx.x;

    while(gid < size){
        atomicAdd(res, data[gid]);
        gid += blockDim.x*blockIdx.x;
    }
}

double calculate_sum_cu(double * data_host, size_t length){
    size_t size = length*sizeof(double);

    double * data_device;
    double * res_device;

    double res_host = 0;

    hipError_t err = hipMalloc((void**) &data_device, size);
    assert(err == 0);
    err = hipMemcpy(data_device, data_host, size, hipMemcpyHostToDevice);
    assert(err == 0);

    err = hipMalloc((void**) &res_device, sizeof(double));
    assert(err == 0);
    err = hipMemcpy(res_device, &res_host, sizeof(double), hipMemcpyHostToDevice);
    assert(err == 0);

    calculate_sum_cu_device<<64, 64>>(data_device, length, res_device);
    err = hipGetLastError();
    assert(err == 0);


    err = hipMemcpy(&res_host, res_device, sizeof(double), hipMemcpyDeviceToHost);
    assert(err == 0);

    hipFree(res_device);
    hipFree(data_device);

    return res_host;
}

